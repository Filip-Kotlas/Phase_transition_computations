#include "hip/hip_runtime.h"
#include "Problem.hpp"

#define COMPUTE_PHASE
#define COMPUTE_CONCENTRATION

#define INIT 7

#define P_INIT INIT


#define C_INIT INIT
/*
*  20 - Fourier along x axis
*  30 - Fourier along y axis
*/

#define C_BOUND 3
/*
*  0 - Dirichlet everywhere
*  1 - Neumann along x, Dirichlet along y
*  2 - Dirichlet along x, Neumann along y
*  3 - Neumann everywhere
*  4 - Dirichlet everywhere with c = 0.007
*/

#define P_BOUND 1
/*
*  0 - Dirichlet
*  1 - Neumann
*/

//#define C_TEST_X

#ifdef C_TEST_X
#define C_INIT 20
#define C_BOUND 1
#endif

#ifdef C_TEST_Y
#define C_INIT 30
#define C_BOUND 2
#endif

#define FORCE 2
/*
*  0 - Force equal 40
*  1 - Force inversely proportional to the distance from the middle
*  2 - Force for zirconium model
*/

Problem::Problem(Parameters param)
:  sizeX(param.sizeX),
   sizeY(param.sizeY),
   domain(param.domain),
   hx((domain.x_right - domain.x_left)/(sizeX-1)),
   hy((domain.y_right - domain.y_left)/(sizeY-1)),
   alpha(param.alpha),
   par_a(param.par_a),
   par_b(param.par_b),
   par_d(param.par_d),
   T(param.T),
   ksi(param.ksi),
   model(param.model)
{
}

Index Problem::getDegreesOfFreedom()
{
    return 2 * this->sizeX * this->sizeY;
}

__cuda_callable__
void Problem::set_rhs_at(const VectorView& u, VectorView& fu, Index i, Index j)
{
    Index offset = this->sizeX * this->sizeY;
    fu[j*this->sizeX + i] = get_rhs_phase_at(u, i, j);
    fu[offset + j*this->sizeX + i] = get_rhs_concentration_at(u, i, j);
}

bool Problem::writeSolution(const Real &t, Index step, const VectorView& u, const std::filesystem::path& output_folder)
{
   /****
    * Filename with step index
    */
   std::stringstream str;
   str << "Result-" << std::setw( 5 ) << std::setfill( '0' ) << step << ".txt";
   std::filesystem::path file_path = output_folder / str.str();

   /****
    * Open file
    */
   std::fstream file;
   file.open( file_path, std::fstream::out | std::fstream::trunc );
   if( ! file )
   {
      std::cerr << "Unable to open the file " << str.str() << std::endl;
      return false;
   }

   /****
    * Write solution
    */
   file << std::scientific << std::setprecision(15);
   for( Index j = 0; j < sizeY; j++ )
   {
      for( Index i = 0; i < sizeX; i++ )
      {
         file << domain.x_left + i * hx << " " << domain.y_left + j * hy << " "
              << phase_at(u, i, j) << " " << conc_at(u, i, j);
         file << std::endl;
      }
      file << std::endl;
   }
   return true;
}

void Problem::set_init_cond_manually(Vector& u, InitialCondition ic)
{
   set_phase_initial_condition(u, ic);
   set_concentration_initial_condition(u, ic);
}

void Problem::set_phase_initial_condition(Vector& u, InitialCondition ic)
{
    Real r = (domain.x_right - domain.x_left)/6;
    Real r1 = r - 0.5*ksi;
    Real r2 = r1 + ksi;

    switch(ic) {
        case InitialCondition::HyperbolicTangent:
            u.forElements( 0, sizeX*sizeY,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    Index j = ind / sizeX;
                    Real radius = sqrt(pow(i*hx - (domain.x_right - domain.x_left)/2, 2) + pow(j*hy - (domain.y_right-domain.y_left)/2, 2));
                    value = 1.0/2 * tanh(-3/ksi*(radius - r1)) + 1.0/2; //needs to be changed according to the correct phases
                } );
            break;

        case InitialCondition::LinearByParts:
            u.forElements( 0, sizeX*sizeY,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    Index j = ind / sizeX;
                    Real radius = sqrt(pow(i*hx - (domain.x_right - domain.x_left)/2, 2) + pow(j*hy - (domain.y_right-domain.y_left)/2, 2));
                    if( radius < r1 )
                    {
                        value = constants::p_alpha;
                    }
                    else if( radius < r2 )
                    {
                        value = constants::p_alpha - (constants::p_alpha - constants::p_beta)*(radius - r1) / (r2 - r1);
                    }
                    else
                    {
                        value = constants::p_beta;
                    }
                } );
            break;
        
        case InitialCondition::ConstantCircle:
            u.forElements( 0, sizeX*sizeY,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    Index j = ind / sizeX;
                    Real radius = sqrt(pow(i*hx - (domain.x_right - domain.x_left)/2, 2) + pow(j*hy - (domain.y_right-domain.y_left)/2, 2));
                    if( radius < r1 )
                    {
                        value = constants::p_alpha;
                    }
                    else
                    {
                        value = constants::p_beta;
                    }
                } );
            break;
        
        case InitialCondition::ConstantHalves:
            u.forElements( 0, sizeX*sizeY,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    //Index j = ind / sizeX;
                    if( i < sizeX/2 )
                    {
                        value = constants::p_alpha;
                    }
                    else
                    {
                        value = constants::p_beta;
                    }
                } );
            break;

        case InitialCondition::Stripe:
            u.forElements( 0, sizeX*sizeY,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    Index j = ind / sizeX;
                    if( i*hx < 0.2 )
                    {
                        value = constants::p_alpha;
                    }
                    else
                    {
                        value = constants::p_beta;
                    }
                } );
            break;

        case InitialCondition::TwoBumps:
            u.forElements( 0, sizeX*sizeY,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    Index j = ind / sizeX;
                    Real y = 2*j*hy;
                    if( (y < 1 && i*hx < y*y*(1-2*y+y*y)/0.625+0.1) ||
                        (y >= 1 && i*hx < (y-1)*(y-1)*(1-2*(y-1)+(y-1)*(y-1))/0.625+0.1))
                    {
                        value = constants::p_alpha;
                    }
                    else
                    {
                        value = constants::p_beta;
                    }
                } );
            break;
        
        case InitialCondition::Star:
            u.forElements( 0, sizeX*sizeY,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    Index j = ind / sizeX;
                    Real radius = sqrt(pow(i*hx - (domain.x_right - domain.x_left)/2, 2) + pow(j*hy - (domain.y_right-domain.y_left)/2, 2));
                    Real phi = atan( (j*hy - (domain.y_right-domain.y_left)/2) / (i*hx - (domain.x_right - domain.x_left)/2));
                    if ((i*hx - (domain.x_right - domain.x_left)/2) < 0 )
                        phi = phi + M_PI;
                    else if ( (i*hx - (domain.x_right - domain.x_left)/2) > 0 && (j*hy - (domain.y_right-domain.y_left)/2) < 0)
                        phi = phi + 2 * M_PI;
                    
                    if ( radius < 0.15 + 0.1 * sin(6 * phi) )
                    {
                        value = constants::p_alpha;
                    }
                    else
                    {
                        value = constants::p_beta;
                    }
                } );
            break;

            case InitialCondition::Perpendicular_Stripes:
                u.forElements( 0, sizeX*sizeY,
                    [ = ] __cuda_callable__( Index ind, Real & value )
                    {
                        Index i = ind % sizeX;
                        Index j = ind / sizeX;
                        if ( (i*hx < 0.1) || (j*hy < 0.1 ) )
                        {
                            value = constants::p_alpha;
                        }
                        else
                        {
                            value = constants::p_beta;
                        }
                    } );
                break;
    }   
}

void Problem::set_concentration_initial_condition(Vector& u, InitialCondition ic)
{
    Index offset = sizeX * sizeY;
    Real r = (domain.x_right - domain.x_left)/6;
    Real r1 = r - 0.5*ksi;
    Real r2 = r1 + ksi;

    switch(ic) {
        case InitialCondition::HyperbolicTangent:
            u.forElements( sizeX*sizeY, sizeX*sizeY*2,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    Index j = ind / sizeX - sizeY;
                    Real radius = sqrt(pow(i*hx - (domain.x_right - domain.x_left)/2, 2) + pow(j*hy - (domain.y_right-domain.y_left)/2, 2));
                    value = constants::c_init_alpha; // TODO: Nutno předělat je to špatně.
                });
            break;
        
        case InitialCondition::LinearByParts:
            u.forElements( sizeX*sizeY, sizeX*sizeY*2,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    Index j = ind / sizeX - sizeY;
                    Real radius = sqrt(pow(i*hx - (domain.x_right - domain.x_left)/2, 2) + pow(j*hy - (domain.y_right-domain.y_left)/2, 2));
                    if( radius < r1 )
                    {
                        value = constants::c_init_alpha;
                    }
                    else if( radius < r2 )
                    {
                        value = constants::c_init_alpha - (constants::c_init_alpha - constants::c_init_beta)*(radius - r1) / (r2 - r1);
                    }
                    else
                    {
                        value = constants::c_init_beta;
                    }
                } );
            break;

        case InitialCondition::ConstantCircle:
            u.forElements( sizeX*sizeY, sizeX*sizeY*2,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    Index j = ind / sizeX - sizeY;
                    Real radius = sqrt(pow(i*hx - (domain.x_right - domain.x_left)/2, 2) + pow(j*hy - (domain.y_right-domain.y_left)/2, 2));
                    if( radius < r )
                    {
                        value = constants::c_init_alpha;
                    }
                    else
                    {
                        value = constants::c_init_beta;
                    }
                } );
            break;
        
        case InitialCondition::ConstantHalves:
            u.forElements( sizeX*sizeY, sizeX*sizeY*2,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    //Index j = ind / sizeX - sizeY;
                    if( i < sizeX/2 )
                    {
                        value = constants::c_init_alpha;
                    }
                    else
                    {
                        value = constants::c_init_beta;
                    }
                } );
            break;
        
        case InitialCondition::Stripe:
            u.forElements( sizeX*sizeY, sizeX*sizeY*2,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    Index j = ind / sizeX - sizeY;
                    if( i*hx < 0.2 )
                    {
                        value = constants::c_init_alpha;
                    }
                    else
                    {
                        value = constants::c_init_beta;
                    }
                } );
            break;
        
        case InitialCondition::TwoBumps:
            u.forElements( sizeX*sizeY, sizeX*sizeY*2,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    Index j = ind / sizeX - sizeY;
                    Real y = 2*j*hy;
                    if( (y < 1 && i*hx < y*y*(1-2*y+y*y)/0.625+0.1) ||
                        (y >= 1 && i*hx < (y-1)*(y-1)*(1-2*(y-1)+(y-1)*(y-1))/0.625+0.1))
                    {
                        value = constants::c_init_alpha;
                    }
                    else
                    {
                        value = constants::c_init_beta;
                    }
                } );
            break;

        case InitialCondition::Star:
            u.forElements( sizeX*sizeY, sizeX*sizeY*2,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    Index j = ind / sizeX - sizeY;
                    Real radius = sqrt(pow(i*hx - (domain.x_right - domain.x_left)/2, 2) + pow(j*hy - (domain.y_right-domain.y_left)/2, 2));
                    Real phi = atan( (j*hy - (domain.y_right-domain.y_left)/2) / (i*hx - (domain.x_right - domain.x_left)/2));
                    if ((i*hx - (domain.x_right - domain.x_left)/2) < 0 )
                        phi = phi + M_PI;
                    else if ( (i*hx - (domain.x_right - domain.x_left)/2) > 0 && (j*hy - (domain.y_right-domain.y_left)/2) < 0)
                        phi = phi + 2 * M_PI;
                    
                    if ( radius < 0.15 + 0.1 * sin(6 * phi) )
                    {
                        value = constants::c_init_alpha;
                    }
                    else
                    {
                        value = constants::c_init_beta;
                    }
                } );
            break;
        
        case InitialCondition::FourierX:
            u.forElements( sizeX*sizeY, sizeX*sizeY*2,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    Index j = ind / sizeX - sizeY;
                    value = 0;
                    for(Index n = 1; n < 5; n++)
                    {
                        Real C_n = pow(1.0/2, n);
                        Real lambda_n = pow(n*M_PI/(domain.x_right-domain.x_left), 2);
                        value += C_n * sin(sqrt(lambda_n) * (i*hx));
                    }
                } );
            break;

        case InitialCondition::FourierY:
            u.forElements( sizeX*sizeY, sizeX*sizeY*2,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    Index j = ind / sizeX - sizeY;
                    value = 0;
                    for(Index n = 1; n < 5; n++)
                    {
                        Real C_n = pow(1.0/2, n);
                        Real lambda_n = pow(n*M_PI/(domain.y_right-domain.y_left), 2);
                        value += C_n * sin(sqrt(lambda_n) * (j*hy));
                    }
                } );
            break;

        case InitialCondition::Perpendicular_Stripes:
            u.forElements( sizeX*sizeY, sizeX*sizeY*2,
                [ = ] __cuda_callable__( Index ind, Real & value )
                {
                    Index i = ind % sizeX;
                    Index j = ind / sizeX - sizeY;
                    if ( (i*hx < 0.1) || (j*hy < 0.1 ) )
                    {
                        value = constants::c_init_alpha;
                    }
                    else
                    {
                        value = constants::c_init_beta;
                    }
                } );
            break;
   }
}

bool Problem::set_init_cond_from_file(Vector& u, const std::filesystem::path& filename)
{
    std::ifstream file(filename);
    if (!file)
    {
        std::cerr << "Nelze otevřít soubor: " << filename << '\n';
        return false;
    }

    Vector host_u(getDegreesOfFreedom());

    std::string line;
    Index count = 0;

    while (std::getline(file, line)) {
        if (line.empty()) continue; // přeskoč prázdné řádky

        std::istringstream iss(line);
        Real x, y, p, c;
        if (!(iss >> x >> y >> p >> c)) {
            std::cout << "Chyba při čtení řádku: " << line << '\n';
            return false;
        }

        if (count >= sizeX * sizeY) {
            std::cout << "Soubor obsahuje více dat než očekáváno.\n";
            return false;
        }

        host_u[count] = p;
        host_u[sizeX * sizeY + count] = c;
        ++count;
    }

    if (count < sizeX * sizeY) {
        std::cout << "Soubor obsahuje méně dat než očekáváno.\n";
        return false;
    }

    u = host_u;

   return true;
}

__cuda_callable__
void Problem::apply_boundary_condition_xdir(Index ind, VectorView u, VectorView fu)
{
    apply_phase_boundary_condition_xdir(ind, u, fu);
    apply_concentration_boundary_condition_xdir(ind, u, fu);
}

__cuda_callable__
void Problem::apply_boundary_condition_ydir(Index ind, VectorView u, VectorView fu)
{
    apply_phase_boundary_condition_ydir(ind, u, fu);
    apply_concentration_boundary_condition_ydir(ind, u, fu);
}

__cuda_callable__
void Problem::apply_phase_boundary_condition_xdir(Index ind, VectorView u, VectorView fu)
{
    // Horní a dolní hrana
    fu[ind] = 0;
    fu[(sizeY - 1) * sizeX + ind] = 0;

    #if P_BOUND == 0  // Dirichlet
    u[ind] = 0;
    u[(sizeY - 1) * sizeX + ind] = 0;

    #elif P_BOUND == 1  // Neumann
    u[ind] = u[ind + sizeX];
    u[(sizeY - 1) * sizeX + ind] = u[(sizeY - 2) * sizeX + ind];
    #endif
}

__cuda_callable__
void Problem::apply_phase_boundary_condition_ydir(Index ind, VectorView u, VectorView fu)
{
    // Levá a pravá hrana
    fu[ind * sizeX] = 0;
    fu[(ind + 1) * sizeX - 1] = 0;

    #if P_BOUND == 0  // Dirichlet
    u[ind * sizeX] = 0;
    u[(ind + 1) * sizeX - 1] = 0;

    #elif P_BOUND == 1  // Neumann
    u[ind * sizeX] = u[ind * sizeX + 1];
    u[(ind + 1) * sizeX - 1] = u[(ind + 1) * sizeX - 2];
    #endif
}

__cuda_callable__
void Problem::apply_concentration_boundary_condition_xdir(Index ind, VectorView u, VectorView fu)
{
    Index offset = sizeX * sizeY;

    fu[offset + ind] = 0;
    fu[offset + (sizeY - 1) * sizeX + ind] = 0;

    #if C_BOUND == 0 || C_BOUND == 2
    u[offset + ind] = 0;
    u[offset + (sizeY - 1) * sizeX + ind] = 0;

    #elif C_BOUND == 1 || C_BOUND == 3
    u[offset + ind] = u[offset + ind + sizeX];
    u[offset + (sizeY - 1) * sizeX + ind] = u[offset + (sizeY - 2) * sizeX + ind];

    #elif C_BOUND == 4
    u[offset + ind] = constants::c_init_beta;
    u[offset + (sizeY - 1) * sizeX + ind] = constants::c_init_beta;
    #endif
}

__cuda_callable__
void Problem::apply_concentration_boundary_condition_ydir(Index ind, VectorView u, VectorView fu)
{
    Index offset = sizeX * sizeY;

    fu[offset + ind * sizeX] = 0;
    fu[offset + (ind + 1) * sizeX - 1] = 0;

    #if C_BOUND == 0 || C_BOUND == 1
    u[offset + ind * sizeX] = 0;
    u[offset + (ind + 1) * sizeX - 1] = 0;

    #elif C_BOUND == 2 || C_BOUND == 3
    u[offset + ind * sizeX] = u[offset + ind * sizeX + 1];
    u[offset + (ind + 1) * sizeX - 1] = u[offset + (ind + 1) * sizeX - 2];

    #elif C_BOUND == 4
    u[offset + ind * sizeX] = constants::c_init_beta;
    u[offset + (ind + 1) * sizeX - 1] = constants::c_init_beta;
    #endif
}


/*
void Problem::apply_boundary_condition(double *u, double *fu)
{
   apply_phase_boundary_condition(u, fu);
   apply_concentration_boundary_condition(u, fu);
}

void Problem::apply_phase_boundary_condition(double *u, double *fu)
{
   for(int i = 0; i < this->sizeX; i++)
   {
      //Dirichlet
      #if P_BOUND == 0
      fu[i] = 0;
      fu[(sizeY-1)*sizeX + i] = 0;

      //Neumann
      #elif P_BOUND == 1
      u[i] = u[i + sizeX];
      u[(sizeY-1)*sizeX + i] = u[(sizeY-2)*sizeX + i];
      fu[i] = 0;
      fu[(sizeY-1)*sizeX + i] = 0;
      #endif
   }
   for(int j = 1; j < this->sizeY-1; j++)
   {
      //Dirichlet
      #if P_BOUND == 0
      fu[j*sizeX] = 0;
      fu[(j+1)*sizeX - 1] = 0;

      //Neumann
      #elif P_BOUND == 1
      u[j*sizeX] = u[j*sizeX + 1];
      u[(j+1)*sizeX - 1] = u[(j+1)*sizeX - 2];
      fu[j*sizeX] = 0;
      fu[(j+1)*sizeX - 1] = 0;
      #endif
   }
}

void Problem::apply_concentration_boundary_condition(double *u, double *fu)
{
   int offset = sizeY * sizeX;

   //Boundary conditions along x direction
   for(int i = 0; i < this->sizeX; i++)
   {
      //Dirichlet 0
      #if C_BOUND == 0 || C_BOUND == 2
      u[offset + i] = 0;
      u[offset + (sizeY-1)*sizeX + i] = 0;
      fu[offset + i] = 0;
      fu[offset + (sizeY-1)*sizeX + i] = 0;
      
      //Neumann
      #elif C_BOUND == 1 || C_BOUND == 3
      u[offset + i] = u[offset + i + sizeX];
      u[offset + (sizeY-1)*sizeX + i] = u[offset + (sizeY-2)*sizeX + i];
      fu[offset + i] = 0;
      fu[offset + (sizeY-1)*sizeX + i] = 0;

      //Dirichlet with c_init_beta
      #elif C_BOUND == 4
      u[offset + i] = constants::c_init_beta;
      u[offset + (sizeY-1)*sizeX + i] = constants::c_init_beta;
      fu[offset + i] = constants::c_init_beta;
      fu[offset + (sizeY-1)*sizeX + i] = constants::c_init_beta;
      #endif
   }

   //Boundary conditions along y direction
   for(int j = 0; j < this->sizeY; j++)
   {
      //Dirichlet 0
      #if C_BOUND == 0 || C_BOUND == 1
      u[offset + j*sizeX] = 0;
      u[offset + (j+1)*sizeX - 1] = 0;
      fu[offset + j*sizeX] = 0;
      fu[offset + (j+1)*sizeX - 1] = 0;

      //Neumann
      #elif C_BOUND == 2 || C_BOUND == 3
      u[offset + j*sizeX] = u[offset + j*sizeX + 1];
      u[offset + (j+1)*sizeX - 1] = u[offset + (j+1)*sizeX - 2];
      fu[offset + j*sizeX] = 0;
      fu[offset + (j+1)*sizeX - 1] = 0;
      
      //Dirichlet with c_init_beta
      #elif C_BOUND == 4
      u[offset + j*sizeX] = constants::c_init_beta;
      u[offset + (j+1)*sizeX - 1] = constants::c_init_beta;
      fu[offset + j*sizeX] = constants::c_init_beta;
      fu[offset + (j+1)*sizeX - 1] = constants::c_init_beta;
      #endif
   }

}

void Problem::apply_concentration_physical_condition(double *u)
{
   for(int i = 0; i < this->sizeX; i++)
   {
      for(int j = 0; j < this->sizeY; j++)
      {
         // Check if the concentration is in allowed range of (c_min, c_max).
         if(u[sizeX*sizeY + j*sizeX + i] < constants::c_min)
         {
            u[sizeX*sizeY + j*sizeX + i] = constants::c_min;
         }
         else if (u[sizeX*sizeY + j*sizeX + i] > constants::c_max)
         {
            u[sizeX*sizeY + j*sizeX + i] = constants::c_max;
         }
      }
   }
}
*/

__cuda_callable__
Real Problem::get_rhs_phase_at(const VectorView& u, Index i, Index j)
{
    return 1.0/alpha * (laplace(u, i, j) + f_0(u, i , j) / ksi / ksi - par_b/ksi*grade_4_polynom(u, i, j)*F(u, i, j));
}

__cuda_callable__
Real Problem::get_rhs_concentration_at(const VectorView& u, Index i, Index j)
{
    return div_D_grad_concentration(u, i, j) + div_D_grad_phase(u, i, j);
}

__cuda_callable__
Real Problem::laplace(const VectorView& u, Index i, Index j)
{
   return (phase_at(u, i - 1, j) - 2*phase_at(u, i, j) + phase_at(u, i + 1, j))/hx/hx +
          (phase_at(u, i, j - 1) - 2*phase_at(u, i, j) + phase_at(u, i, j + 1))/hy/hy;
}

__cuda_callable__
Real Problem::div_D_grad_concentration(const VectorView& u, Index i, Index j)
{
	Real coeff_plus_half = (get_conc_diff_coef(u, i + 1, j) + get_conc_diff_coef(u, i, j)) / 2;
	Real coeff_minus_half = (get_conc_diff_coef(u, i, j) + get_conc_diff_coef(u, i - 1, j)) / 2;
	Real x_direction = coeff_plus_half * (conc_at(u, i+1, j) - conc_at(u, i, j)) / hx
				         - coeff_minus_half * (conc_at(u, i, j) - conc_at(u, i-1, j)) / hx;

	coeff_plus_half = (get_conc_diff_coef(u, i, j+1) + get_conc_diff_coef(u, i, j)) / 2;
	coeff_minus_half = (get_conc_diff_coef(u, i, j) + get_conc_diff_coef(u, i, j-1)) / 2;
	Real y_direction = coeff_plus_half * (conc_at(u, i, j+1) - conc_at(u, i, j)) / hy
                      	 - coeff_minus_half * (conc_at(u, i, j) - conc_at(u, i, j-1)) / hy;

	return x_direction / hx + y_direction / hy;
}

__cuda_callable__
Real Problem::div_D_grad_phase(const VectorView& u, Index i, Index j)
{
	Real coeff_plus_half = (get_phas_diff_coef(u, i + 1, j) + get_phas_diff_coef(u, i, j)) / 2;
	Real coeff_minus_half = (get_phas_diff_coef(u, i, j) + get_phas_diff_coef(u, i - 1, j)) / 2;
	Real x_direction = coeff_plus_half * (phase_at(u, i+1, j) - phase_at(u, i, j)) / hx
				         - coeff_minus_half * (phase_at(u, i, j) - phase_at(u, i-1, j)) / hx;

	coeff_plus_half = (get_phas_diff_coef(u, i, j+1) + get_phas_diff_coef(u, i, j)) / 2;
	coeff_minus_half = (get_phas_diff_coef(u, i, j) + get_phas_diff_coef(u, i, j-1)) / 2;
	Real y_direction = coeff_plus_half * (phase_at(u, i, j+1) - phase_at(u, i, j)) / hy
                      	 - coeff_minus_half * (phase_at(u, i, j) - phase_at(u, i, j-1)) / hy;

	return x_direction / hx + y_direction / hy;
}

__cuda_callable__
Real Problem::get_conc_diff_coef(const VectorView& u, Index i, Index j)
{
    return par_b*par_d/30.0
            * conc_at(u, i, j)
            * (1 - conc_at(u, i, j))
            * constants::M_Nb_beta(T)
		    * pow(constants::M_Nb_alpha(T)/constants::M_Nb_beta(T), polynom_p(u, i, j))
		    * sec_deriv_of_g_w_resp_to_c(u, i, j);
}

__cuda_callable__
Real Problem::get_phas_diff_coef(const VectorView& u, Index i, Index j)
{
   return par_b*par_d/30.0
          * conc_at(u, i, j)
		    * (1 - conc_at(u, i, j))
          * constants::M_Nb_beta(T)
		    * pow(constants::M_Nb_alpha(T)/constants::M_Nb_beta(T), polynom_p(u, i, j))
		    * deriv_of_g_w_resp_to_c_and_p(u, i, j);
}

__cuda_callable__
Real Problem::f_0(const VectorView& u, Index i, Index j)
{
   return par_a*phase_at(u, i, j)*(1 - phase_at(u, i, j))*(phase_at(u, i, j) - 1.0/2.0);
}

__cuda_callable__
Real Problem::F(const VectorView& u, Index i, Index j)
{
   #if FORCE == 0
   return 20;

   #elif FORCE == 1
   Real mid_x = (domain.x_right - domain.x_left)/2;
   Real mid_y = (domain.y_right - domain.y_left)/2;
   Real r = sqrt(pow(i*hx - mid_x, 2) + pow(j*hy - mid_y, 2));
   return -2/std::max(r, 0.1);

   #elif FORCE == 2
   return constants::G_m_alpha(conc_at(u, i, j), T) - constants::G_m_beta(conc_at(u, i, j), T);
   
   #endif
}

/*
double Problem::G(const double &t, double *u, int i, int j)
{
   double coefficients[5] = {0, 1.0/2, 0, 1.0/4, 0};
   double G = 0; 
   for(int n = 1; n < 5; n++)
   {
      double F_n = coefficients[n];
      
      #ifdef C_TEST_X
      double lambda_n = pow(n*M_PI/(domain.x_right-domain.x_left), 2);
      G += F_n * sin(sqrt(lambda_n) * (i*hx));
      #endif

      #ifdef C_TEST_Y
      double lambda_n = pow(n*M_PI/(domain.y_right-domain.y_left), 2);
      G += F_n * sin(sqrt(lambda_n) * (j*hy));
      #endif
   }
   return G;
}
*/

__cuda_callable__
Real Problem::grade_4_polynom(const VectorView& u, Index i, Index j)
{
   return pow(phase_at(u, i, j), 2) * pow(phase_at(u, i, j) - 1.0, 2);
}

__cuda_callable__
Real Problem::polynom_p(const VectorView& u, Index i, Index j)
{
   return 6*pow(phase_at(u, i, j), 5) - 15*pow(phase_at(u, i, j), 4) + 10*pow(phase_at(u, i, j), 3);
}

__cuda_callable__
Real Problem::der_polynom_p(const VectorView& u, Index i, Index j)
{
   return 30*(pow(phase_at(u, i, j), 4) - 2*pow(phase_at(u, i, j), 3) + pow(phase_at(u, i, j), 2));
}

__cuda_callable__
Real Problem::sec_deriv_of_g_w_resp_to_c(const VectorView& u, Index i, Index j)
{
   Real c = conc_at(u, i, j);
   Real d2_G_alpha_wrt_c = constants::R * T / (c*(1-c))
   							 - 2 * constants::L_0_alpha;
   Real d2_G_beta_wrt_c = constants::R * T / (c*(1-c))
   							 - 2 * constants::L_0_beta(T)
							 + (6 - 12*c) * constants::L_0_i_beta(T);
   return polynom_p(u, i, j)*d2_G_alpha_wrt_c + (1 - polynom_p(u, i, j))*d2_G_beta_wrt_c;
}

__cuda_callable__
Real Problem::deriv_of_g_w_resp_to_c_and_p(const VectorView& u, Index i, Index j)
{
   Real c = conc_at(u, i, j);
   Real d_G_alpha_wrt_c = constants::G_Nb_alpha_0(T)
                            - constants::G_Zr_alpha_0(T)
                            + constants::R * T * (log(c) - log(1-c))
                            + (1 - 2*c) * constants::L_0_alpha;
   Real d_G_beta_wrt_c = constants::G_Nb_beta_0(T)
						   - constants::G_Zr_beta_0(T)
						   + constants::R * T * (log(c) - log(1-c))
						   + (1 - 2 * c) * constants::L_0_beta(T)
						   + (6*c - 6*pow(c, 2) - 1) * constants::L_0_i_beta(T);

   return der_polynom_p(u, i, j)*(d_G_alpha_wrt_c - d_G_beta_wrt_c);
}