#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <iostream>
#include <vector>
#include <filesystem>
#include <json.hpp>
#include <string>

#include "Problem.hpp"
#include "Parameters.hpp"
#include "types.hpp"
#include "InitialCondition.hpp"

int main(int argc, char** argv) {
    Parameters parameters = Parameters::load("config/config.json");
    std::filesystem::path result_path = "Results";
    if (argc == 2)
        result_path = argv[1];

    std::filesystem::path info_path   = result_path / "info";
    std::filesystem::path setup_path  = info_path / "parameters.txt";
    std::filesystem::path calc_path   = result_path / "calculations";
    std::filesystem::path config_path = info_path / "config.json";
    std::filesystem::path latex_path = info_path / "parameters.tex";
    
    try {
        std::filesystem::create_directories(result_path);
        std::filesystem::create_directories(info_path);
        std::filesystem::create_directories(calc_path);
    }
    catch (const std::filesystem::filesystem_error& e) {
        std::cerr << "Error creating folders: " << e.what() << std::endl;
        return EXIT_FAILURE;
    }

    parameters.save_human_readable(setup_path);
    parameters.save_copy_of_config("config/config.json", config_path);
    parameters.save_for_latex(latex_path);

    Problem problem(parameters);
    Vector u(problem.getDegreesOfFreedom());

    if (!parameters.init_cond_from_file ||
        !problem.set_init_cond_from_file(u, calc_path / parameters.init_cond_file_path)) {
        std::cout << "Setting initial condition by code." << std::endl;
        InitialCondition init_cond(parameters.init_condition, parameters.domain, parameters.sizeX, parameters.sizeY, parameters.ksi);
        problem.set_init_cond_manually(u, init_cond);
    }

    ODESolver solver;
    solver.setTau( parameters.integrationTimeStep );
    solver.setTime( parameters.initial_time );
    
    Index step_number = static_cast<Index>(round(solver.getTime() / parameters.timeStep));
    problem.writeSolution(0.0, step_number, u, calc_path);
    
    while( solver.getTime() < parameters.final_time ) {
        solver.setStopTime( TNL::min( solver.getTime() + parameters.timeStep, parameters.final_time ) );

        auto rhs = [ = ] __cuda_callable__( const TNL::Containers::StaticArray< 2, Index >& ind, const VectorView& u, VectorView& fu ) mutable {
            problem.set_rhs_at(u, fu, ind.x(), ind.y());
        };

        auto boundary_xdir = [ = ] __cuda_callable__( const Index ind, VectorView& u, VectorView& fu ) mutable {
            problem.apply_boundary_condition_xdir(ind , u, fu);
        };

        auto boundary_ydir = [ = ] __cuda_callable__( const Index ind, VectorView& u, VectorView& fu ) mutable {
            problem.apply_boundary_condition_ydir(ind , u, fu);
        };

        auto time_stepping = [ = ]( const Real& t, const Real& tau, const VectorView& u, VectorView& fu )
        {
            // iterate over inner points only
            TNL::Containers::StaticArray< 2, Index > begin = {1, 1};
            TNL::Containers::StaticArray< 2, Index > end = {parameters.sizeX - 1, parameters.sizeY - 1};
            TNL::Algorithms::parallelFor< Device >(begin, end, rhs, u, fu );

            // iterate over boundary points
            TNL::Algorithms::parallelFor< Device >(0, parameters.sizeX, boundary_xdir, u, fu );
            TNL::Algorithms::parallelFor< Device >(0, parameters.sizeY, boundary_ydir, u, fu );
        };
        auto stepStart = std::chrono::high_resolution_clock::now();
        solver.solve( u, time_stepping );
        auto stepEnd = std::chrono::high_resolution_clock::now();
        step_number = static_cast<Index>(round(solver.getTime() / parameters.timeStep));
        problem.writeSolution(solver.getTime(), step_number, u, calc_path);

        double time_per_step = (std::chrono::duration<double>(stepEnd - stepStart).count());

        double remaining_time = time_per_step * (parameters.frame_num * parameters.final_time
                                                 / (parameters.final_time - parameters.initial_time)
                                                 - step_number);
      
        int hours = static_cast<int>(remaining_time) / 3600;
        int minutes = (static_cast<int>(remaining_time) % 3600) / 60;
        int seconds = remaining_time - (hours * 3600 + minutes * 60);
        
        std::cout << "Steps completed: " << step_number << " / " << parameters.frame_num << " => " << std::fixed
                  << std::setprecision(2) << ( double ) step_number / ( double ) parameters.frame_num * 100.0 << "% ";
        std::cout << "     Time remaining: " 
                  << hours << "h " << minutes << "m " << seconds << "s"
                  << std::endl;
    }
}