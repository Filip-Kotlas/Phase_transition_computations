#include "hip/hip_runtime.h"
#include "Parameters.hpp"
#include <fstream>
#include <iomanip>
#include <sstream>
#include <iostream>
#include <cmath>
#include <algorithm>

using nlohmann::json;

Parameters Parameters::load(const std::filesystem::path& filename) {
    std::ifstream file(filename);
    if (!file) {
        throw std::runtime_error("Unable to open the file " + filename.string());
    }

    json j;
    file >> j;

    Parameters p;

    auto solver = j.at("solver");
    p.initial_time  = solver.value("initial_time", 0.0);
    p.final_time    = solver.value("final_time", 0.3);
    p.sizeX         = solver.value("sizeX", 200);
    p.sizeY         = solver.value("sizeY", 200);
    p.frame_num     = solver.value("frame_num", 100);
    p.timeStep      = (p.final_time - p.initial_time) / p.frame_num;

    auto domain = solver.at("domain");
    p.domain.x_left  = domain.value("x_left", -1.0);
    p.domain.x_right = domain.value("x_right", 1.0);
    p.domain.y_left  = domain.value("y_left", -1.0);
    p.domain.y_right = domain.value("y_right", 1.0);

    /*
    int model_value = solver.value("model", 1);
    if (!(model_value >= 1 && model_value <= 4)) {
        throw std::runtime_error("Neplatná hodnota 'model' v JSON.");
    }
    p.model = static_cast<MODEL>(model_value);
    */
   
    double computed_integration_step = pow(std::min((p.domain.x_right - p.domain.x_left) / (p.sizeX - 1),
                                                    (p.domain.y_right - p.domain.y_left) / (p.sizeY - 1)),
                                           2) / 5.0;

    if (solver.value("custom_integration_time_step", false)) {
        p.integrationTimeStep = solver.value("integration_time_step", computed_integration_step);
    } else {
        p.integrationTimeStep = computed_integration_step;
    }

    p.init_cond_from_file = solver.value("init_cond_from_file", false);
    p.init_cond_file_path = solver.value("init_cond_file_path", "");

    std::string ic_str = solver.value("initial_condition", "hyperbolic_tangent");
    
    if (ic_str == "hyperbolic_tangent")         p.init_condition = ICType::HyperbolicTangent;
    else if (ic_str == "linear_by_parts")       p.init_condition = ICType::LinearByParts;
    else if (ic_str == "constant_circle")       p.init_condition = ICType::ConstantCircle;
    else if (ic_str == "constant_halves")       p.init_condition = ICType::ConstantHalves;
    else if (ic_str == "stripe")                p.init_condition = ICType::Stripe;
    else if (ic_str == "two_bumps")             p.init_condition = ICType::TwoBumps;
    else if (ic_str == "three_bumps")           p.init_condition = ICType::ThreeBumps;
    else if (ic_str == "star")                  p.init_condition = ICType::Star;
    else if (ic_str == "perpendicular_stripes") p.init_condition = ICType::PerpendicularStripes;
    else if (ic_str == "box")                   p.init_condition = ICType::Box;
    else if (ic_str == "random_bumps")          p.init_condition = ICType::RandomBumps;
    else throw std::runtime_error("Unknown initial_condition in config: " + ic_str);

    auto problem = j.at("problem");
    p.alpha = problem.value("alpha", 1.0);
    p.beta  = problem.value("beta", 1.0);
    p.par_a = problem.value("a", 1.0);
    p.par_b = problem.value("b", 0.1);
    p.par_d = problem.value("d", 5e15);
    p.T = problem.value("T", 1200);
    p.ksi   = problem.value("ksi", 0.01);
    
    return p;
}

void Parameters::save_human_readable(const std::filesystem::path& filename) const {
    std::ofstream file(filename);
    if (!file) {
        throw std::runtime_error("Unable to open the file " + filename.string());
    }

    file << std::left << std::setw(24) << "Initial time:" << std::right << std::setw(28) << initial_time << std::endl;
    file << std::left << std::setw(24) << "Final time:"   << std::right << std::setw(28) << final_time << std::endl;

    std::ostringstream oss;
    oss << "[(" << std::fixed << std::setprecision(2) 
        << domain.x_left << ", " << domain.x_right << ")"
        << "(" << domain.y_left << ", " << domain.y_right << ")]";
    file << std::left << std::setw(24) << "Domain:" << std::right << std::setw(28) << oss.str() << std::endl;

    file << std::left << std::setw(24) << "SizeX:"  << std::right << std::setw(28) << sizeX << std::endl;
    file << std::left << std::setw(24) << "SizeY:"  << std::right << std::setw(28) << sizeY << std::endl;
    file << std::left << std::setw(24) << "Time step:" << std::right << std::setw(28) << timeStep << std::endl;
    file << std::left << std::setw(24) << "Integration time step:" << std::right << std::setw(28) << integrationTimeStep << std::endl;
    file << std::left << std::setw(24) << "Alpha:" << std::right << std::setw(28) << alpha << std::endl;
    file << std::left << std::setw(24) << "Beta:"  << std::right << std::setw(28) << beta << std::endl;
    file << std::left << std::setw(24) << "Par_a:" << std::right << std::setw(28) << par_a << std::endl;
    file << std::left << std::setw(24) << "Par_b:" << std::right << std::setw(28) << par_b << std::endl;
    file << std::left << std::setw(24) << "Par_d:" << std::right << std::setw(28) << par_d << std::endl;
    file << std::left << std::setw(24) << "T:" << std::right << std::setw(28) << T << std::endl;
    file << std::left << std::setw(24) << "Ksi:"   << std::right << std::setw(28) << ksi << std::endl;
    //file << std::left << std::setw(24) << "Model:" << std::right << std::setw(28) << static_cast<int>(model) << std::endl;
}

void Parameters::save_for_latex(const std::filesystem::path& filename) const {
    std::ofstream file(filename);
    if (!file) {
        throw std::runtime_error("Unable to open the file " + filename.string());
    }

    file << "\\textbf{Oblast:}" << std::endl << std::endl;
    file << "\\begin{tabular}{ll}" << std::endl;
    file << "\\(\\Omega\\) & \\((" << std::defaultfloat 
    << domain.x_left << ", " << domain.x_right << ") \\times "
    << "(" << domain.y_left << ", " << domain.y_right << ")\\) \\\\" << std::endl;
    file << "\\(N_x\\) & " << sizeX << " \\\\" << std::endl;
    file << "\\(N_y\\) & " << sizeY << " \\\\" << std::endl;
    file << "\\end{tabular}" << std::endl << std::endl;
    file << "\\textbf{Časové parametry:}" << std::endl << std::endl;
    file << "\\begin{tabular}{ll}" << std::endl;
    file << "\\(t_{0}\\) & " << initial_time << " \\\\" << std::endl;
    file << "\\(t_{max}\\) & " << final_time << " \\\\" << std::endl;
    file << "\\(\\tau\\) & " << integrationTimeStep << " \\\\" << std::endl;
    file << "\\end{tabular}" << std::endl << std::endl;
    file << "\\textbf{Parametry simulace:}" << std::endl << std::endl;
    file << "\\begin{tabular}{ll}" << std::endl;
    file << "\\(\\alpha\\) & " << alpha << " \\\\" << std::endl;
    file << "\\(\\beta\\) & " << beta << " \\\\" << std::endl;
    file << "\\(a\\) & " << par_a << " \\\\" << std::endl;
    file << "\\(b\\) & " << par_b << " \\\\" << std::endl;
    file << "\\(d\\) & " << par_d << " \\\\" << std::endl;
    file << "\\(T\\) & " << T << " \\\\" << std::endl;
    file << "\\(\\xi\\) & " << ksi << " \\\\" << std::endl;
    //file << "Model & " << static_cast<int>(  model) << " \\\\" << std::endl;
    file << "\\end{tabular}" << std::endl;
}

void Parameters::save_copy_of_config(const std::filesystem::path& original_path,
                                     const std::filesystem::path& copy_path) const {
    std::ifstream src(original_path, std::ios::binary);
    if (!src) {
        throw std::runtime_error("Unable to open original config: " + original_path.string());
    }
    std::ofstream dst(copy_path, std::ios::binary);
    if (!dst) {
        throw std::runtime_error("Unable to open original config: " + copy_path.string());
    }
    dst << src.rdbuf();
}